#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2015 by                                                 *
 *   WARSAW UNIVERSITY OF TECHNOLOGY                                       *
 *   FACULTY OF PHYSICS                                                    *
 *   NUCLEAR THEORY GROUP                                                  *
 *   See also AUTHORS file                                                 *
 *                                                                         *
 *   This file is a part of GPE for GPU project.                           *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/ 
#include <math.h>
#include <complex.h>      // not std::complex!


#include "reductions.cuh"
#include "gpe_engine.cuh"


    
/***************************************************************************/ 
/******************************** GLOBALS **********************************/
/***************************************************************************/


gpe_mem_t gpe_mem;
gpe_flags_t gpe_flags;


// ============================ CONSTANT MEMORY ALLOCATION ==================================================

/*
 * TODO: check available size of constant memory
 *       think how to "dynamically allocate constant memory" - in runtime
 */
// device constants
__constant__ double d_alpha;
__constant__ double d_beta;
__constant__ double d_qfcoeff; // quantum friction coeff
__constant__ cuCplx d_step_coeff; // 0.5*dt/(i*alpha-beta)

// vortex properties
__constant__ double d_vortex_x0;
__constant__ double d_vortex_y0;
__constant__ int8_t d_vortex_Q;

// reciprocal lattice constants
__constant__ double d_kkx[NX];
__constant__ double d_kky[NY];
__constant__ double d_kkz[NZ];
__constant__ cuCplx d_exp_kkx2[NX]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * kx^2 )
__constant__ cuCplx d_exp_kky2[NY]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * ky^2 )
__constant__ cuCplx d_exp_kkz2_over_nxyz[NZ]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * kz^2 ) / nxyz


#ifndef MAX_USER_PARAMS
#define MAX_USER_PARAMS 32
#endif
__constant__ double d_user_param[MAX_USER_PARAMS];
__constant__ uint d_nx; // lattice size in x direction
__constant__ uint d_ny; // lattice size in y direction
__constant__ uint d_nz; // lattice size in z direction
__constant__ double d_dt;
__constant__ double d_t0;
__constant__ double d_npart;

/***************************************************************************/ 
/****************************** FUNCTIONS **********************************/
/***************************************************************************/

// =========================== Lattice ========================================================================

void gpe_get_lattice(int *_nx, int *_ny, int *_nz)
{
    *_nx = nx;
    *_ny = ny;
    *_nz = nz;
}

/*
 * Allocates memory and creates arrays containing reciprocal lattice points' coordinates.
 */
inline gpe_result_t gpe_reciprocal_lattice_init( double alpha, double beta)
{
    /* ***************************************************************************************
     * TODO:
     *      - ask if c3/(GAMMA*GAMMA) could not be change to simplier form (NOT IMPORTANT ...)
     *      - think of adding OpenMP sections (probably not parallel for!)
     */
    
    /* NOTE : nx , ny , nz = 2j forall j integers (e.g. even numbers for the lattice dimensions) */
    // Initialize lattice in momentum space (first Brullion zone)
    /* initialize the k-space lattice */
    const double dt = gpe_mem.dt;
    uint ui;
    int i,j;
    double r;
    
    // Generate arrays on host
    gpemalloc(gpe_mem.kkx,nx,double);
    gpemalloc(gpe_mem.kky,ny,double);
    gpemalloc(gpe_mem.kkz,nz,double);  
    
    for ( i = 0 ; i <= nx / 2 - 1 ; i++ ) {
        gpe_mem.kkx[ i ] = 2. * ( double ) M_PI / nx * ( double ) i ;  }
    j = - i ;
    for ( i = nx / 2 ; i < nx ; i++ ) 
    {
        gpe_mem.kkx[ i ] = 2. * ( double ) M_PI / nx * ( double ) j ; 
        j++ ;
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kkx), gpe_mem.kkx, nx*sizeof(double)) ) ;

    for ( i = 0 ; i <= ny / 2 - 1 ; i++ ) {
        gpe_mem.kky[ i ] = 2. * ( double ) M_PI / ny * ( double ) i ;  }
    j = - i ;
    for ( i = ny / 2 ; i < ny ; i++ ) 
    {
        gpe_mem.kky[ i ] = 2. * ( double ) M_PI / ny * ( double ) j ; 
        j++ ;
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kky), gpe_mem.kky, ny*sizeof(double)) ) ;

    for ( i = 0 ; i <= nz / 2 - 1 ; i++ ) {
        gpe_mem.kkz[ i ] = 2. * ( double ) M_PI / nz * ( double ) i ;  }
    j = - i ;
    for ( i = nz / 2 ; i < nz ; i++ ) 
    {
        gpe_mem.kkz[ i ] = 2. * ( double ) M_PI / nz * ( double ) j ; 
        j++ ;
    }    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kkz), gpe_mem.kkz, nz*sizeof(double)) ) ;
    
    // 0.5*dt/(i*alpha-beta)*GAMMA
    cplx c1=GAMMA*0.5*dt + I*0.0;
    cplx c2=-1.0*beta + I*alpha;
    cplx c3=c1/c2;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_step_coeff), &c3, sizeof(cuCplx)) ) ;
    
    // kinetic operator mulipliers
    cuCplx *carr;
    
    // nx direction
    gpemalloc(carr,nx,cuCplx);
    for(ui=0; ui<nx; ui++)
    {
        c1=cexp(c3*gpe_mem.kkx[ui]*gpe_mem.kkx[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkx2), carr, nx*sizeof(cuCplx)) ) ;
    free(carr);
    
    // ny direction
    gpemalloc(carr,ny,cuCplx);
    for(ui=0; ui<ny; ui++)
    {
        c1=cexp(c3*gpe_mem.kky[ui]*gpe_mem.kky[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kky2), carr, ny*sizeof(cuCplx)) ) ;
    free(carr);
    
    // nz direction
    gpemalloc(carr,nz,cuCplx);
    for(ui=0; ui<nz; ui++)
    {
        c1=cexp(c3*gpe_mem.kkz[ui]*gpe_mem.kkz[ui]/(GAMMA*GAMMA)) / (double)(nxyz); // NOTE: here we divide to 
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkz2_over_nxyz), carr, nz*sizeof(cuCplx)) ) ;
    free(carr);
    
    return GPE_SUCCES;
}

/*
 * Needed when changing evolution type without rectreating whole engine.
 */
inline gpe_result_t gpe_reciprocal_lattice_change( double alpha, double beta)
{
    /* ***************************************************************************************
     * TODO:
     *      - ask if c3/(GAMMA*GAMMA) could not be change to simplier form (NOT IMPORTANT ...)
     *      - think of adding OpenMP sections (probably not parallel for!)
     */
    
    double dt = gpe_mem.dt;
    uint ui;
    int i,j;
    double r;
    
    // 0.5*dt/(i*alpha-beta)*GAMMA
    cplx c1=GAMMA*0.5*dt + I*0.0;
    cplx c2=-1.0*beta + I*alpha;
    cplx c3=c1/c2;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_step_coeff), &c3, sizeof(cuCplx)) ) ;
    
    // kinetic operator mulipliers
    cuCplx *carr;
    
    // nx direction
    gpemalloc(carr,nx,cuCplx);
    for(ui=0; ui<nx; ui++)
    {
        c1=cexp(c3*gpe_mem.kkx[ui]*gpe_mem.kkx[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkx2), carr, nx*sizeof(cuCplx)) ) ;
    free(carr);
    
    // ny direction
    gpemalloc(carr,ny,cuCplx);
    for(ui=0; ui<ny; ui++)
    {
        c1=cexp(c3*gpe_mem.kky[ui]*gpe_mem.kky[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kky2), carr, ny*sizeof(cuCplx)) ) ;
    free(carr);
    
    // nz direction
    gpemalloc(carr,nz,cuCplx);
    for(ui=0; ui<nz; ui++)
    {
        c1=cexp(c3*gpe_mem.kkz[ui]*gpe_mem.kkz[ui]/(GAMMA*GAMMA)) / (double)(nxyz); // NOTE: here we divide to 
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkz2_over_nxyz), carr, nz*sizeof(cuCplx)) ) ;
    free(carr);
    
    return GPE_SUCCES;
}


// =========================== User interface ========================================================================

int gpe_create_engine(double alpha, double beta, double dt, double npart, int nthreads)
{
    uint ui;
    int i,j;
    double r;
    gpe_result_t res;
    
    #ifndef GAMMA
        return -99; // not supported mode
    #endif
    
    // Set flags
    gpe_flags.vortex_set = 0;
    
    // Set number of blocks, if number of threads is given
    gpe_mem.threads=nthreads;
    gpe_mem.blocks=(int)ceil((float)nxyz/nthreads);

//     printf("GPU SETTING: THREADS=%d, BLOCKS=%d, THREADS*BLOCKS=%d, nxyz=%d\n",gpe_mem.threads,gpe_mem.blocks,gpe_mem.threads*gpe_mem.blocks,nxyz);
    
    // Fill const memory
    ui=nx;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_nx), &ui, sizeof(uint)) ) ;
    ui=ny;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_ny), &ui, sizeof(uint)) ) ;
    ui=nz;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_nz), &ui, sizeof(uint)) ) ;   
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(double)) ) ;
    gpe_mem.alpha=alpha;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double)) ) ;
    gpe_mem.beta=beta;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(double)) ) ;
    gpe_mem.dt=dt;
    r=0.0;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &r, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &r, sizeof(double)) ) ;
    gpe_mem.t0=0.0;
    gpe_mem.it=0;
    gpe_mem.qfcoeff=0.0;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_npart), &npart, sizeof(double)) ) ;
    gpe_mem.npart=npart;
    
    // create reciprocal lattice (in bonduary of first Brullion zone)
    res = gpe_reciprocal_lattice_init(alpha, beta);
    
    
    
    // TODO: Create separate function for this and probably create array of plans...
    // create cufft plans
    hipfftResult cufft_result;
    cufft_result=hipfftCreate(&gpe_mem.plan); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    cufft_result=hipfftSetAutoAllocation(gpe_mem.plan, 0); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    size_t workSize;
    cufft_result=hipfftMakePlan3d(gpe_mem.plan, nx, ny, nz, HIPFFT_Z2Z, &workSize);
    if(workSize<sizeof(cuCplx)*nxyz) workSize=sizeof(cuCplx)*nxyz;
    cuErrCheck( hipMalloc( &gpe_mem.d_wrk , workSize ) );
    cufft_result=hipfftSetWorkArea(gpe_mem.plan, gpe_mem.d_wrk); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    
    // allocate memory for workspace on device
    cuErrCheck( hipMalloc( &gpe_mem.d_wrk2, sizeof(cuCplx)*nxyz ) );
    cuErrCheck( hipMalloc( &gpe_mem.d_psi,  sizeof(cuCplx)*nxyz ) );
    cuErrCheck( hipMalloc( &gpe_mem.d_psi2, sizeof(cuCplx)*nxyz ) );
    gpe_mem.d_wrk2R = (double *) gpe_mem.d_wrk2; 
    
    gpe_mem.d_wrk3R = NULL;
    gpe_mem.d_wrk3C = NULL;
    
#ifdef DIPOLAR
    // TODO: Check if it is not necessary!
    //cuErrCheck( hipMalloc( &gpe_mem.d_dipolar_wrk, sizeof(cuCplx)*nxyz) );
#endif
    
    return GPE_SUCCES; // success
}

int gpe_destroy_engine()
{
    
    hipfftResult cufft_result;
    free(gpe_mem.kkx);
    free(gpe_mem.kky);
    free(gpe_mem.kkz);
    cufft_result=hipfftDestroy(gpe_mem.plan); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    cuErrCheck( hipFree(gpe_mem.d_wrk) );
    cuErrCheck( hipFree(gpe_mem.d_wrk2) );
    cuErrCheck( hipFree(gpe_mem.d_psi) );
    cuErrCheck( hipFree(gpe_mem.d_psi2) );
    if(gpe_mem.d_wrk3R != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3R) );
    if(gpe_mem.d_wrk3C != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3C) );
    
    return GPE_SUCCES; // success
}

int gpe_change_alpha_beta(double alpha, double beta)
{
    
    uint ui;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(double)) ) ;
    gpe_mem.alpha=alpha;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double)) ) ;
    gpe_mem.beta=beta;
    
    // update reciprocal lattice
    gpe_reciprocal_lattice_change(alpha, beta);
    
    return 0;
}

int gpe_set_rte_evolution()
{
    gpe_change_alpha_beta(1.0,0.0);
    return GPE_SUCCES;
}

int gpe_set_ite_evolution()
{
    gpe_change_alpha_beta(0.0,1.0);
    return GPE_SUCCES;
}

int gpe_set_time(double t0)
{
    
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &t0, sizeof(double)) ) ;
    gpe_mem.t0=t0;
    gpe_mem.it=0;    
    
    return 0;
}

int gpe_set_user_params(int size, double *params)
{
    if(size>MAX_USER_PARAMS) return -9;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_user_param), params, MAX_USER_PARAMS*sizeof(double)) );
    
    return 0;
}

int gpe_set_quantum_friction_coeff(double qfcoeff)
{
    
    if(qfcoeff!=0.0)
    {
        qfcoeff=qfcoeff/( GAMMA*(double)(nxyz) );
        cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &qfcoeff, sizeof(double)) ) ;
        gpe_mem.qfcoeff=qfcoeff;
        
        if(gpe_mem.d_wrk3R==NULL) cuErrCheck( hipMalloc( &gpe_mem.d_wrk3R , sizeof(double)*nxyz ) );
        if(gpe_mem.d_wrk3C==NULL) cuErrCheck( hipMalloc( &gpe_mem.d_wrk3C , sizeof(cuCplx)*nxyz ) );
    }
    else
    {
        cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &qfcoeff, sizeof(double)) ) ;
        gpe_mem.qfcoeff=qfcoeff;
        
        if(gpe_mem.d_wrk3R != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3R) );
        if(gpe_mem.d_wrk3C != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3C) );   
        
        gpe_mem.d_wrk3R = NULL;
        gpe_mem.d_wrk3C = NULL;
    }
    
    return 0;
}


// ======================= Quantum vortices interface =======================================================

int gpe_set_vortex(const double vortex_x0, const double vortex_y0, const int8_t Q) 
{
    hipError_t err;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_x0), &vortex_x0, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_y0), &vortex_y0, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_Q), &Q, sizeof(int8_t)) ) ;
    
    gpe_flags.vortex_set = 1;
    
    return 0; // success
}

/*
 * This function imprints vortex parallel to z axis crossing x,y plane in (x0,y0) point
 * double d_vortex_x0, d_vortex_y0 - position of vortex in xy plane
 * uint8_t d_Q_vortex - topological charge of vortex
 * NOTE: It is considered that x0 and y0 should be chosen out of lattice points in case 
 *       phase is corectly (mathematically) defined in every lattice point (check atan2).
 */
__global__ void __gpe_imprint_vortexline_zdir_(cuCplx *psi)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi = psi[ixyz];
    double abs_psi, phase;
    double _x,_y;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        
        _x = constgpu(ix) - 1.0*(NX/2) - d_vortex_x0;
        _y = constgpu(iy) - 1.0*(NY/2) - d_vortex_y0;
        //_iz = constgpu(iz) - 1.0*(NZ/2);
        
        //abs_psi = sqrt(lpsi.x*lpsi.x + lpsi.y*lpsi.y);
        abs_psi = hypot(lpsi.x, lpsi.y);
        phase = atan2(_x,_y); // atan2(0,0) == -pi/2
        phase *= (double) (d_vortex_Q);
        //if (d_vortex_Q != 1) phase *= (double) (d_vortex_Q);
        lpsi.x = abs_psi*cos(phase);
        lpsi.y = abs_psi*sin(phase);
        
        psi[ixyz] = lpsi;
    }
}

// ======================= Density/Normalization ============================================================

// TODO: Test speed with cublas

/**
 * Function computes density from wave function psi
 * */
inline __device__  double gpe_density(cuCplx psi)
{
    return GAMMA * (psi.x*psi.x + psi.y*psi.y); // |psi|^2 * GAMMA, where GAMMA=1 for particles, GAMMA=2 for dimers
}

__global__ void __gpe_compute_density__(cuCplx *psi_in, double *rho_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        rho_out[ixyz] = gpe_density(psi_in[ixyz]);
    }
}

/**
 * Computes density and saves in array of complex numbers (as real part).
 * Suitable for dipolar interactions.
 * */
__global__ void __gpe_compute_density2C__(cuCplx *psi_in, cuCplx *rho_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        rho_out[ixyz].x = gpe_density(psi_in[ixyz]);
        rho_out[ixyz].y = 0.;
    }
}

__global__ void __gpe_normalize__(cuCplx *psi_inout, double *sumrho)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
//         if(ixyz==0) printf("sumrho[0]=%f\n", sumrho[0]);
        psi_inout[ixyz] = cplxScale(psi_inout[ixyz], sqrt(d_npart/sumrho[0]));
    }
}

// Normalizes wavefunction
int gpe_normalize(cuCplx *psi, double *wrk)
{
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk);
    cuErrCheck( local_reduction(wrk, nxyz, wrk, gpe_mem.threads, 0) );
    __gpe_normalize__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk);
    
    return GPE_SUCCES;
}

int gpe_normalize_psi()
{
    return gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R);
}

static inline int gpe_normalize_psi(double *chemical_potential)
{    
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    cuErrCheck( local_reduction(gpe_mem.d_wrk2R, nxyz, gpe_mem.d_wrk2R, gpe_mem.threads, 0) );
    __gpe_normalize__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    if (chemical_potential)
    {
        double norm;
        cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R, sizeof(double), hipMemcpyDeviceToHost) ); 
        *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
    }
    
    return GPE_SUCCES;
}

int gpe_get_density(double *t, double * density)
{
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    cuErrCheck( hipMemcpy( density , gpe_mem.d_wrk2R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );
    
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    return 0;
}


// =================== Accesing wavefunction =============================================================================

int gpe_set_psi(double t, cuCplx * psi)
{
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &t, sizeof(double)) ) ;
    gpe_mem.it=0;
    gpe_mem.t0=t;
    cuErrCheck( hipMemcpy( gpe_mem.d_psi , psi , sizeof(cuCplx)*nxyz, hipMemcpyHostToDevice ) );
    
    return 0;
}

int gpe_get_psi(double *t, cuCplx * psi)
{
    
    cuErrCheck( hipMemcpy( psi , gpe_mem.d_psi , sizeof(cuCplx)*nxyz, hipMemcpyDeviceToHost ) );
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    return 0;
}

// ======================= Evolution algorithm =============================================================================

/**
 * construct  exp(-i*dt*V/2) and apply exp(-i*dt*V/2) * psi 
 * */
__global__ void __gpe_exp_Vstep1_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        lrho = gpe_density(lpsi); // compute density
        lv=gpe_external_potential(ix, iy, iz, it) + gpe_dEDFdn(lrho,it); // external potential + mean field
        
        wrkR[ixyz]=lv; // it will be use later
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save
    }
}

__global__ void __gpe_exp_Vstep1_qf_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, double *qfpotential)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        lrho = gpe_density(lpsi); // compute density
        lv=gpe_external_potential(ix, iy, iz, it) + gpe_dEDFdn(lrho,it) + qfpotential[ixyz]; 
           // external potential + mean field + quantum friction potential
        
        wrkR[ixyz]=lv; // it will be use later
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save
    }    
}

__global__ void __gpe_exp_Vstep2_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        lpsi=cplxMul(lpsi, exp_lv); // finalize step from predictor
        
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1)); // external potential + mean field - take average
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep2_part1_(cuCplx *psi_in, cuCplx *psi_out, double * wrkR)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    
    // registers
    cuCplx exp_lv;

    if(ixyz<nxyz)
    {
        exp_lv = cplxExp( cplxScale(d_step_coeff, wrkR[ixyz]) );
        psi_out[ixyz]=cplxMul(psi_in[ixyz], exp_lv); // finalize step from predictor     
    }    
}

__global__ void __gpe_exp_Vstep2_part2_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1)); // external potential + mean field - take average
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep2_part2_qf_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC, double *qfpotential)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1) + qfpotential[ixyz]) ; 
          // external potential + mean field + quantum friction potential - take average 
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep3_(cuCplx *psi_inout, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    
    if(ixyz<nxyz)
    {
        psi_inout[ixyz] = cplxMul(psi_inout[ixyz], wrkC[ixyz]); // apply and save      
    }    
}

__global__ void __gpe_multiply_by_expT__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    cuCplx _wavef;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        _wavef=psi_in[ixyz]; // bring to register
        _wavef=cplxMul(_wavef,d_exp_kkx2[ix]);
        _wavef=cplxMul(_wavef,d_exp_kky2[iy]);
        _wavef=cplxMul(_wavef,d_exp_kkz2_over_nxyz[iz]); // note - normalization factor is included here
        psi_out[ixyz]=_wavef; // send to global memory
    }    
}

__global__ void __gpe_multiply_by_k2_qf__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_qfcoeff*( d_kkx[ix]*d_kkx[ix] + d_kky[iy]*d_kky[iy] + d_kkz[iz]*d_kkz[iz] ) ); 
            
    }    
}

__global__ void __gpe_overlap_imag_qf__(cuCplx *psi1, cuCplx *psi2, double *overlap)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    double lrho;
    cuCplx lpsi;
    if(ixyz<nxyz)
    {
        lpsi = psi1[ixyz]; // psi to register
        lrho = gpe_density(lpsi); // compute density
        overlap[ixyz]= cplxMulI( cplxConj(lpsi),  psi2[ixyz] )/(lrho+GPE_QF_EPSILON);
    }
}

int gpe_compute_qf_potential(cuCplx *psi, cuCplx *wrk, double *qfpotential)
{
    hipfftResult cufft_result;
    
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, psi, wrk, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_k2_qf__<<<gpe_mem.blocks, gpe_mem.threads>>>(wrk, wrk);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, wrk, wrk, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_overlap_imag_qf__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk, qfpotential);

        
    return 0;
}


// ===================== Evolution interface ===========================================

/**
 * Function evolves wave funcion nt steps 
 * */
int gpe_evolve(int nt)
{
    printf("Function not implemented!\n");
    return GPE_SUCCES;
}

/**
 * Function evolves wave funcion nt steps 
 * */
int gpe_evolve_qf(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
                
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return 0;
}

// TODO: Check if this works better!
int gpe_evolve_vortex(int nt)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        // changing the phase
        __gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
        cuErrCheck( hipGetLastError() );
        
        
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
                
                // changing the phase
                __gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
                cuErrCheck( hipGetLastError() );
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return 0;
}


// ========================= Energy counting =================================================

__global__ void __gpe_compute_vext__(uint it, double *rho, double *wrk)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        wrk[ixyz]=rho[ixyz]*gpe_external_potential(ix, iy, iz, it);
    }
}

__global__ void __gpe_compute_vint__(uint it, double *rho, double *wrk)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        wrk[ixyz]=gpe_EDF(rho[ixyz], it);
    }
}

__global__ void __gpe_compute_vext_vint__(uint it, double *rho, double *wrk1, double *wrk2)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    double lrho;
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        lrho=rho[ixyz];
        wrk1[ixyz]=lrho*gpe_external_potential(ix, iy, iz, it);
        wrk2[ixyz]=gpe_EDF(lrho, it);
    }
}

__global__ void __gpe_multiply_by_k2__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], ( d_kkx[ix]*d_kkx[ix] + d_kky[iy]*d_kky[iy] + d_kkz[iz]*d_kkz[iz] )/(constgpu(2.0*GAMMA*nxyz)) ); 
            
    }    
}

__global__ void __gpe_overlap_real__(cuCplx *psi1, cuCplx *psi2, double *overlap)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        overlap[ixyz]= cplxMulR( cplxConj(psi1[ixyz]),  psi2[ixyz] );
    }
}

int gpe_energy(double *t, double *ekin, double *eint, double *eext)
{
    int ierr;
    hipfftResult cufft_result;
   
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    if(gpe_mem.beta==0.0) // normalize - otherwise is normalized every time step
    {
        ierr=gpe_normalize_psi();
        if(ierr!=0) return ierr;
    }
    
    // Compute density
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    // Compute <V_ext> and <V_int>
    double * wrk1 = (double *)gpe_mem.d_wrk;
    double * wrk2 = wrk1 + nxyz;
    __gpe_compute_vext_vint__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_wrk2R, wrk1, wrk2);
    cuErrCheck( local_reduction(wrk1, nxyz, wrk1, gpe_mem.threads, 0) );
    cuErrCheck( local_reduction(wrk2, nxyz, wrk2, gpe_mem.threads, 0) );
    cuErrCheck( hipMemcpy( eext , wrk1 , sizeof(double), hipMemcpyDeviceToHost ) );
    cuErrCheck( hipMemcpy( eint , wrk2 , sizeof(double), hipMemcpyDeviceToHost ) );
        
    // Compute <T> - kinetic energy
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi2, HIPFFT_FORWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    __gpe_multiply_by_k2__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
    cuErrCheck( local_reduction(gpe_mem.d_wrk2R, nxyz, gpe_mem.d_wrk2R, gpe_mem.threads, 0) );
    cuErrCheck( hipMemcpy( ekin , gpe_mem.d_wrk2R , sizeof(double), hipMemcpyDeviceToHost ) );
    
    return 0;
}

// ========================== Currents of probability ================================

__global__ void __gpe_multiply_by_kx__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kkx[ix]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

__global__ void __gpe_multiply_by_ky__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kky[iy]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

__global__ void __gpe_multiply_by_kz__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kkz[iz]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

int gpe_get_currents(double *t, double * jx, double * jy, double * jz)
{
    int ierr;
    
    hipfftResult cufft_result;
   
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    int alloc=0;
    if(gpe_mem.d_wrk3R==NULL) // I need extra memory
    {
        alloc=1;
        cuErrCheck( hipMalloc( &gpe_mem.d_wrk3R , sizeof(double)*nxyz ) );
    }
    
    // move to momentum space
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi2, HIPFFT_FORWARD);  
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    
    // Compute d / dx and jx
    __gpe_multiply_by_kx__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jx , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );    
    
    // Compute d / dy and jy
    __gpe_multiply_by_ky__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jy , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );
    
    // Compute d / dz and jz
    __gpe_multiply_by_kz__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jz , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );

    // Free memory
    if(alloc) 
    {
        cuErrCheck( hipFree(gpe_mem.d_wrk3R) );  
        gpe_mem.d_wrk3R = NULL;      
    }
        
    return GPE_SUCCES;
}




// ===================== Dipolar evolution specific functions ======================================================

#ifdef DIPOLAR
/*
 * We denote:
 * vdd  - dipole-dipole interaction pseudopotential, must be convolved with density of wavefunction.
 * vdip - dipolar inteactions contribution to mean-field pseudopotential
 * vcon - contact inteactions contribution to mean-field pseudopotential
 * vint - vdip + vcon
 * 
 * suffix: _k - variable/function in reciprocal space
 */


/*
 * This function computes total internal interactions propagator's exponent
 * TODO: Use this when whole interactions term defined in Fourier space
 * 
 * Uses density Fourier transform and multiplies it by Fourier transform of particle-particle interaction potential.
 * NOTE: density_k and vint_k_out can be phisically the same arrays!
 */
__global__ void __gpe_comupute_vint_k__(cuCplx* density_k, cuCplx* vint_k_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    cuCplx _vint;
    
    if (ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        
        _vint = cplxScale( density_k[ixyz], gpe_vint_k(d_kkx[ix],d_kky[iy],d_kkz[iz])/((double) nxyz) ); // NOTE: normalization factor for CUFFT included here
        vint_k_out[i] = _vint;
    }
}

/*
 * This function computes total internal interactions propagator's exponent
 * 
 * Uses Fourier transform of density and multiplies it by Fourier transform of particle-particle interaction potential.
 * NOTE: density_k and vint_k_out can be phisically the same arrays!
 * 
 * @param density_k - Fourier transform of density of wavefunction
 * @param vdip_k_out - potential of dipolar part of interactions in Fourier space (make inverse Fourier transform to get real-space function)
 */
__global__ void __gpe_compute_vdip_k__(cuCplx* density_k, cuCplx* vdip_k_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    cuCplx _vdip;
    
    if (ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        
        // multipling fourier transform of dipole-dipole interaction potential and fourier transform of density
        _vdip = cplxScale( density_k[ixyz], gpe_vdd_k(d_kkx[ix],d_kky[iy],d_kkz[iz])/((double) nxyz) ); // NOTE: normalization factor for CUFFT included here
        vdip_k_out[ixyz] = _vdip;
    }
}

/**
 * For Strang splitting.
 * Constructs  exp(-i*dt*V/2) and applies exp(-i*dt*V/2) * psi , where  V = (Vext + Vcon + Vdip).
 * 
 * NOTE: Assuming Vdip is counted and saved in psi_out array before.
 * 
 * */
__global__ void __gpe_dipolar_exp_Vstep__(uint it, cuCplx *psi_in, cuCplx *psi_out, cuCplx *vdip)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv, lvdip;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lvdip = vdip[ixyz];
        lpsi = psi_in[ixyz]; // psi to register
        lpsi = gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        
        lrho = gpe_density(lpsi); // compute density
        lv   = gpe_external_potential(ix, iy, iz, it) + gpe_dEDFdn(lrho,it) + lvdip.x; // external potential + mean field, NOTE: Taking only real part of vdip
        
        //wrkR[ixyz]=lv; // it will be used later
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save
        //printf("x: %d\ty: %d\tz: %d\t\tpsi %e + %ej\n",ix,iy,iz,lpsi.x,lpsi.y);
    }  
}


/* ***************************************** DIPOLAR EVOLUTION ********************************************* */



/**
 * Function evolves wave funcion nt steps with dipolar interactions.
 * Using Strang splitting.
 * TODO: Check if Strang splitting is proper.
 * TODO: Implement quantum friction.
 * 
 * */
int gpe_evolve_dipolar(int nt)
{
    hipfftResult cufft_result;
    int i;
    
    for(i=0; i<nt; i++)
    {
        
        // TODO: Use cufft D2Z and __gpe_compute_density__ and gpe_mem.d_wrk2R (first half)
        // TODO: Think of batched cufft
        
        /* ***  potential part exp(V dt/2) *** */
        // here computes density and saves as real part of array for psi copy array
        __gpe_compute_density2C__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
//          printf("# 1. Printing psi\n");
//          print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi, nxyz);
//          hipDeviceSynchronize();
//          printf("# 2. Printing density\n");
//          print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2,nxyz);
//          hipDeviceSynchronize();
        
        
        // here count CUFFT of density
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
//         printf("# 3. Printing FFT of density\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2, nxyz);
//         hipDeviceSynchronize();
        
        // here multiply fourier transform of density by Vdd
        __gpe_compute_vdip_k__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2,gpe_mem.d_psi2);
//         printf("# 4. Printing vdip in reciprocal space\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2,  nxyz);
//         hipDeviceSynchronize();
        
        // here count CUFFT backward (dipole-dipole interactions' integral)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD); // TODO: Check normalization
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
//         printf("# 5. Printing vdip in real space\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2, nxyz);
//         hipDeviceSynchronize();
        
        // perform exp(V dt/2)
        __gpe_dipolar_exp_Vstep__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi, gpe_mem.d_psi2); // psi_in. psi_out, vdip
//         printf("# 6. Printing psi after exp(V dt/2)\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi, nxyz);
//         hipDeviceSynchronize();
        
        
        
        /* *** kinetic part exp(T dt) *** */
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
//         printf("# 7. Printing psi in reciprocal\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi, nxyz);
//         hipDeviceSynchronize();
        
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        
        
        /* ***  potential part exp(V dt/2) *** */
        // here computes density and saves as real part of array for psi copy array
        __gpe_compute_density2C__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
//         printf("# 8. Printing psi\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi, nxyz);
//         hipDeviceSynchronize();
//         printf("# 9. Printing density\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2, nxyz);
//         hipDeviceSynchronize();
        
        // here count CUFFT of density
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // here multiply fourier transform of density by Vdd
        __gpe_compute_vdip_k__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2,gpe_mem.d_psi2);
//         printf("# 11. Printing vdip in reciprocal space\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi2,  nxyz);
//         hipDeviceSynchronize();
        
        // here count CUFFT backward (dipole-dipole interactions' integral)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD); // TODO: Check normalization
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // perform exp(V dt/2)
        __gpe_dipolar_exp_Vstep__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi, gpe_mem.d_psi2); // psi_in. psi_out, vdip
//         printf("# 12. Printing psi after one step\n");
//         print_gpu_array<<<gpe_mem.blocks, gpe_mem.threads>>>( gpe_mem.d_psi,  nxyz);
//         hipDeviceSynchronize();
        
        
        
        // if ITE evolution is set then normalize
        if(gpe_mem.beta!=0.0)
        {
            cuErrCheck( gpe_normalize_psi() );
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return 0;
}


/**
 * Function evolves wave funcion nt steps with dipolar interactions.
 * Using Strang splitting.
 * TODO: Check if Strang splitting is proper.
 * TODO: Implement quantum friction.
 * 
 * */
int gpe_evolve_dipolar(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        
        // TODO: Use cufft D2Z and __gpe_compute_density__ and gpe_mem.d_wrk2R (first half)
        // TODO: Think of batched cufft
        
        /* ***  potential part exp(V dt/2) *** */
        // computing Vdip
        __gpe_compute_density2C__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);  // here computes density and saves as real part of array for psi copy array
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);         // here count CUFFT of density
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_compute_vdip_k__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2,gpe_mem.d_psi2);     // here multiply fourier transform of density by Vdd
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);         // here count CUFFT backward (dipole-dipole interactions' integral)
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result; // TODO: Check normalization
        
        // perform exp(Vext dt/2)exp(Vcon dt/2)exp(Vdip dt/2)
        __gpe_dipolar_exp_Vstep__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi, gpe_mem.d_psi2); // psi_in. psi_out, vdip
                
        
        /* *** kinetic part exp(T dt) *** */
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);   
        
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        
        
        /* ***  potential part exp(V dt/2) *** */
        // here computes density and saves as real part of array for psi copy array
        __gpe_compute_density2C__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        // here count CUFFT of density
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // here multiply fourier transform of density by Vdd
        __gpe_compute_vdip_k__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2,gpe_mem.d_psi2);
        
        // here count CUFFT backward (dipole-dipole interactions' integral)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD); // TODO: Check normalization
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // perform exp(V dt/2)
        __gpe_dipolar_exp_Vstep__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi, gpe_mem.d_psi2); // psi_in. psi_out, vdip
        
        
        
        // if ITE evolution is set then normalize
        if(gpe_mem.beta!=0.0)
        {
            cuErrCheck( gpe_normalize_psi(chemical_potential) );
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return 0;
}

__global__ void __gpe_compute_vdip_energy__( cuCplx *psi, cuCplx *vdip, double *wrkR)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    double lrho, lvdip;
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        lrho  = gpe_density(psi[ixyz]);
        lvdip = vdip[ixyz].x; // taking only real part
        
        wrkR[ixyz] = lrho * lvdip; 
    }
}

int gpe_energy_dipolar(double *t, double *edip)
{
    hipfftResult cufft_result;
    //int i;
    
    // here count convolution vdd and |psi|^2 = Vdip (operator)
    __gpe_compute_density2C__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
    if (cufft_result!= HIPFFT_SUCCESS) return (int) cufft_result;
    __gpe_compute_vdip_k__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2,gpe_mem.d_psi2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD); // TODO: Check normalization
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    
    double* wrkR = (double*) gpe_mem.d_wrk;
    // here count <psi| Vdip |psi>
    __gpe_compute_vdip_energy__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi,gpe_mem.d_psi2,wrkR);
    
    cuErrCheck( local_reduction(wrkR, nxyz, wrkR, gpe_mem.threads, 0) );
    cuErrCheck( hipMemcpy( edip , wrkR , sizeof(double), hipMemcpyDeviceToHost ) ); // copies only first element
    
    *edip *= .5; // term 1/2 before energy density (summing by pairs of particles)
    
    return GPE_SUCCES;
}


#endif


// ======================================= TESTING ================================================================================

__global__ void print_gpu_array_nans( cuCplx* psi, int size)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if( ixyz<nxyz && ixyz < size)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        ix -= nx/2;
        iy -= ny/2;
        iz -= nz/2;
        
        if (isnan(psi[i].x) || isnan(psi[i].y)) printf("x: %d\ty: %d\tz: %d\t\tpsi %e + %ej\n",ix,iy,iz,psi[i].x,psi[i].y);
    }
}

// ==================================================================================================================================
