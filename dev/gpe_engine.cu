#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2015 by                                                 *
 *   WARSAW UNIVERSITY OF TECHNOLOGY                                       *
 *   FACULTY OF PHYSICS                                                    *
 *   NUCLEAR THEORY GROUP                                                  *
 *   See also AUTHORS file                                                 *
 *                                                                         *
 *   This file is a part of GPE for GPU project.                           *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/ 
#include <math.h>
#include <complex.h>      // not std::complex!


#include "reductions.cuh"
#include "gpe_engine.cuh"


    
/***************************************************************************/ 
/******************************** GLOBALS **********************************/
/***************************************************************************/


gpe_mem_t gpe_mem;
gpe_flags_t gpe_flags;


// ============================ CONSTANT MEMORY ALLOCATION ==================================================

/*
 * TODO: check available size of constant memory
 *       think how to "dynamically allocate constant memory" - in runtime
 */
// device constants
__constant__ double d_alpha;
__constant__ double d_beta;
__constant__ double d_qfcoeff; // quantum friction coeff
__constant__ cuCplx d_step_coeff; // 0.5*dt/(i*alpha-beta)

// vortex properties
__constant__ double d_vortex_x0;
__constant__ double d_vortex_y0;
__constant__ int8_t d_vortex_Q;

// reciprocal lattice constants
__constant__ double d_kkx[NX];
__constant__ double d_kky[NY];
__constant__ double d_kkz[NZ];
__constant__ cuCplx d_exp_kkx2[NX]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * kx^2 )
__constant__ cuCplx d_exp_kky2[NY]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * ky^2 )
__constant__ cuCplx d_exp_kkz2_over_nxyz[NZ]; // exp( (dt/(i*alpha-beta)) * 1/(2gamma) * kz^2 ) / nxyz


#ifndef MAX_USER_PARAMS
#define MAX_USER_PARAMS 32
#endif
__constant__ double d_user_param[MAX_USER_PARAMS];
__constant__ uint d_nx; // lattice size in x direction
__constant__ uint d_ny; // lattice size in y direction
__constant__ uint d_nz; // lattice size in z direction
__constant__ double d_dt;
__constant__ double d_t0;
__constant__ double d_npart;

/***************************************************************************/ 
/****************************** FUNCTIONS **********************************/
/***************************************************************************/

// =========================== Lattice ========================================================================

void gpe_get_lattice(int *_nx, int *_ny, int *_nz)
{
    *_nx = nx;
    *_ny = ny;
    *_nz = nz;
}

/*
 * Allocates memory and creates arrays containing reciprocal lattice points' coordinates.
 */
inline gpe_result_t gpe_reciprocal_lattice_init( double alpha, double beta)
{
    /* ***************************************************************************************
     * TODO:
     *      - ask if c3/(GAMMA*GAMMA) could not be change to simplier form (NOT IMPORTANT ...)
     *      - think of adding OpenMP sections (probably not parallel for!)
     */
    
    /* NOTE : nx , ny , nz = 2j forall j integers (e.g. even numbers for the lattice dimensions) */
    // Initialize lattice in momentum space (first Brullion zone)
    /* initialize the k-space lattice */
    const double dt = gpe_mem.dt;
    uint ui;
    int i,j;
    double r;
    
    // Generate arrays on host
    gpemalloc(gpe_mem.kkx,nx,double);
    gpemalloc(gpe_mem.kky,ny,double);
    gpemalloc(gpe_mem.kkz,nz,double);  
    
    for ( i = 0 ; i <= nx / 2 - 1 ; i++ ) {
        gpe_mem.kkx[ i ] = 2. * ( double ) M_PI / nx * ( double ) i ;  }
    j = - i ;
    for ( i = nx / 2 ; i < nx ; i++ ) 
    {
        gpe_mem.kkx[ i ] = 2. * ( double ) M_PI / nx * ( double ) j ; 
        j++ ;
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kkx), gpe_mem.kkx, nx*sizeof(double)) ) ;

    for ( i = 0 ; i <= ny / 2 - 1 ; i++ ) {
        gpe_mem.kky[ i ] = 2. * ( double ) M_PI / ny * ( double ) i ;  }
    j = - i ;
    for ( i = ny / 2 ; i < ny ; i++ ) 
    {
        gpe_mem.kky[ i ] = 2. * ( double ) M_PI / ny * ( double ) j ; 
        j++ ;
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kky), gpe_mem.kky, ny*sizeof(double)) ) ;

    for ( i = 0 ; i <= nz / 2 - 1 ; i++ ) {
        gpe_mem.kkz[ i ] = 2. * ( double ) M_PI / nz * ( double ) i ;  }
    j = - i ;
    for ( i = nz / 2 ; i < nz ; i++ ) 
    {
        gpe_mem.kkz[ i ] = 2. * ( double ) M_PI / nz * ( double ) j ; 
        j++ ;
    }    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_kkz), gpe_mem.kkz, nz*sizeof(double)) ) ;
    
    // 0.5*dt/(i*alpha-beta)*GAMMA
    cplx c1=GAMMA*0.5*dt + I*0.0;
    cplx c2=-1.0*beta + I*alpha;
    cplx c3=c1/c2;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_step_coeff), &c3, sizeof(cuCplx)) ) ;
    
    // kinetic operator mulipliers
    cuCplx *carr;
    
    // nx direction
    gpemalloc(carr,nx,cuCplx);
    for(ui=0; ui<nx; ui++)
    {
        c1=cexp(c3*gpe_mem.kkx[ui]*gpe_mem.kkx[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkx2), carr, nx*sizeof(cuCplx)) ) ;
    free(carr);
    
    // ny direction
    gpemalloc(carr,ny,cuCplx);
    for(ui=0; ui<ny; ui++)
    {
        c1=cexp(c3*gpe_mem.kky[ui]*gpe_mem.kky[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kky2), carr, ny*sizeof(cuCplx)) ) ;
    free(carr);
    
    // nz direction
    gpemalloc(carr,nz,cuCplx);
    for(ui=0; ui<nz; ui++)
    {
        c1=cexp(c3*gpe_mem.kkz[ui]*gpe_mem.kkz[ui]/(GAMMA*GAMMA)) / (double)(nxyz); // NOTE: here we divide to 
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkz2_over_nxyz), carr, nz*sizeof(cuCplx)) ) ;
    free(carr);
    
    return GPE_SUCCESS;
}

/*
 * Needed when changing evolution type without rectreating whole engine.
 */
inline gpe_result_t gpe_reciprocal_lattice_change( double alpha, double beta)
{
    /* ***************************************************************************************
     * TODO:
     *      - ask if c3/(GAMMA*GAMMA) could not be change to simplier form (NOT IMPORTANT ...)
     *      - think of adding OpenMP sections (probably not parallel for!)
     */
    
    double dt = gpe_mem.dt;
    uint ui;
    int i,j;
    double r;
    
    // 0.5*dt/(i*alpha-beta)*GAMMA
    cplx c1=GAMMA*0.5*dt + I*0.0;
    cplx c2=-1.0*beta + I*alpha;
    cplx c3=c1/c2;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_step_coeff), &c3, sizeof(cuCplx)) ) ;
    
    // kinetic operator mulipliers
    cuCplx *carr;
    
    // nx direction
    gpemalloc(carr,nx,cuCplx);
    for(ui=0; ui<nx; ui++)
    {
        c1=cexp(c3*gpe_mem.kkx[ui]*gpe_mem.kkx[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkx2), carr, nx*sizeof(cuCplx)) ) ;
    free(carr);
    
    // ny direction
    gpemalloc(carr,ny,cuCplx);
    for(ui=0; ui<ny; ui++)
    {
        c1=cexp(c3*gpe_mem.kky[ui]*gpe_mem.kky[ui]/(GAMMA*GAMMA));
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kky2), carr, ny*sizeof(cuCplx)) ) ;
    free(carr);
    
    // nz direction
    gpemalloc(carr,nz,cuCplx);
    for(ui=0; ui<nz; ui++)
    {
        c1=cexp(c3*gpe_mem.kkz[ui]*gpe_mem.kkz[ui]/(GAMMA*GAMMA)) / (double)(nxyz); // NOTE: here we divide to normalize CUFFT
        carr[ui].x=creal(c1); carr[ui].y=cimag(c1);
        //carr[ui] = (cuCplx) c1; // cuCplx and cplx should be binary-compatible
    }
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_exp_kkz2_over_nxyz), carr, nz*sizeof(cuCplx)) ) ;
    free(carr);
    
    return GPE_SUCCESS;
}


// =========================== User interface ========================================================================

int gpe_create_engine(double alpha, double beta, double dt, double npart, int nthreads)
{
    uint ui;
    int i,j;
    double r;
    gpe_result_t res;
    
    // check if mode is right
    #ifndef GAMMA
        return -99; // not supported mode
    #endif
    gpe_check_particle_type();
    gpe_print_interactions_type();
    
    // Set flags
    gpe_flags.vortex_set = 0;
    gpe_flags.phase_set = 0;
    
    // Set number of blocks, if number of threads is given
    gpe_mem.threads=nthreads;
    gpe_mem.blocks=(int)ceil((float)nxyz/nthreads);

//     printf("# GPU SETTING: THREADS=%d, BLOCKS=%d, THREADS*BLOCKS=%d, nxyz=%d\n",gpe_mem.threads,gpe_mem.blocks,gpe_mem.threads*gpe_mem.blocks,nxyz);
    printf("\n");
    
    // Fill const memory
    ui=nx;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_nx), &ui, sizeof(uint)) ) ;
    ui=ny;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_ny), &ui, sizeof(uint)) ) ;
    ui=nz;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_nz), &ui, sizeof(uint)) ) ;   
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(double)) ) ;
    gpe_mem.alpha=alpha;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double)) ) ;
    gpe_mem.beta=beta;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(double)) ) ;
    gpe_mem.dt=dt;
    r=0.0;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &r, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &r, sizeof(double)) ) ;
    gpe_mem.t0=0.0;
    gpe_mem.it=0;
    gpe_mem.qfcoeff=0.0;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_npart), &npart, sizeof(double)) ) ;
    gpe_mem.npart=npart;
    
    // create reciprocal lattice (in bonduary of first Brullion zone)
    res = gpe_reciprocal_lattice_init(alpha, beta);
    
    
    
    // TODO: Create separate function for this and probably create array of plans...
    // create cufft plans
    hipfftResult cufft_result;
    cufft_result=hipfftCreate(&gpe_mem.plan); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    cufft_result=hipfftSetAutoAllocation(gpe_mem.plan, 0); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    size_t workSize;
    cufft_result=hipfftMakePlan3d(gpe_mem.plan, nx, ny, nz, HIPFFT_Z2Z, &workSize);
    if(workSize<sizeof(cuCplx)*nxyz) workSize=sizeof(cuCplx)*nxyz;
    cuErrCheck( hipMalloc( &gpe_mem.d_wrk , workSize ) );
    cufft_result=hipfftSetWorkArea(gpe_mem.plan, gpe_mem.d_wrk); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    
    // allocate memory for workspace on device
    cuErrCheck( hipMalloc( &gpe_mem.d_wrk2, sizeof(cuCplx)*nxyz ) );
    cuErrCheck( hipMalloc( &gpe_mem.d_psi,  sizeof(cuCplx)*nxyz ) );
    cuErrCheck( hipMalloc( &gpe_mem.d_psi2, sizeof(cuCplx)*nxyz ) );
    gpe_mem.d_wrk2R = (double *) gpe_mem.d_wrk2; 
    
    gpe_mem.d_wrk3R = NULL;
    gpe_mem.d_wrk3C = NULL;
    gpe_mem.d_phase = NULL;
    
#ifdef DIPOLAR
    // TODO: Check if it is not necessary!
    //cuErrCheck( hipMalloc( &gpe_mem.d_dipolar_wrk, sizeof(cuCplx)*nxyz) );
#endif
    
    return GPE_SUCCESS; // success
}

int gpe_destroy_engine()
{
    
    hipfftResult cufft_result;
    free(gpe_mem.kkx);
    free(gpe_mem.kky);
    free(gpe_mem.kkz);
    cufft_result=hipfftDestroy(gpe_mem.plan); if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    cuErrCheck( hipFree(gpe_mem.d_wrk) );
    cuErrCheck( hipFree(gpe_mem.d_wrk2) );
    cuErrCheck( hipFree(gpe_mem.d_psi) );
    cuErrCheck( hipFree(gpe_mem.d_psi2) );
    if(gpe_mem.d_wrk3R != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3R) );
    if(gpe_mem.d_wrk3C != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3C) );
    if(gpe_mem.d_phase != NULL) cuErrCheck( hipFree(gpe_mem.d_phase) );
    
    return GPE_SUCCESS; // success
}

int gpe_change_alpha_beta(double alpha, double beta)
{
    
    uint ui;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(double)) ) ;
    gpe_mem.alpha=alpha;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double)) ) ;
    gpe_mem.beta=beta;
    
    // update reciprocal lattice
    gpe_reciprocal_lattice_change(alpha, beta);
    
    return 0;
}

int gpe_set_rte_evolution()
{
    gpe_change_alpha_beta(1.0,0.0);
    return GPE_SUCCESS;
}

int gpe_set_ite_evolution()
{
    gpe_change_alpha_beta(0.0,1.0);
    return GPE_SUCCESS;
}

int gpe_set_time(double t0)
{
    
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &t0, sizeof(double)) ) ;
    gpe_mem.t0=t0;
    gpe_mem.it=0;    
    
    return 0;
}

int gpe_set_user_params(int size, double *params)
{
    if(size>MAX_USER_PARAMS) return -9;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_user_param), params, MAX_USER_PARAMS*sizeof(double)) );
    
    return 0;
}

int gpe_set_quantum_friction_coeff(double qfcoeff)
{
    
    if(qfcoeff!=0.0)
    {
        qfcoeff=qfcoeff/( GAMMA*(double)(nxyz) );
        cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &qfcoeff, sizeof(double)) ) ;
        gpe_mem.qfcoeff=qfcoeff;
        
        if(gpe_mem.d_wrk3R==NULL) cuErrCheck( hipMalloc( &gpe_mem.d_wrk3R , sizeof(double)*nxyz ) );
        if(gpe_mem.d_wrk3C==NULL) cuErrCheck( hipMalloc( &gpe_mem.d_wrk3C , sizeof(cuCplx)*nxyz ) );
    }
    else
    {
        cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_qfcoeff), &qfcoeff, sizeof(double)) ) ;
        gpe_mem.qfcoeff=qfcoeff;
        
        if(gpe_mem.d_wrk3R != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3R) );
        if(gpe_mem.d_wrk3C != NULL) cuErrCheck( hipFree(gpe_mem.d_wrk3C) );   
        
        gpe_mem.d_wrk3R = NULL;
        gpe_mem.d_wrk3C = NULL;
    }
    
    return 0;
}


// ======================= Quantum vortices interface =======================================================

int gpe_set_vortex(const double vortex_x0, const double vortex_y0, const int8_t Q) 
{
    hipError_t err;
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_x0), &vortex_x0, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_y0), &vortex_y0, sizeof(double)) ) ;
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_vortex_Q), &Q, sizeof(int8_t)) ) ;
    
    gpe_flags.vortex_set = 1;
    
    return 0; // success
}

/*
 * This function imprints vortex parallel to z axis crossing x,y plane in (x0,y0) point
 * double d_vortex_x0, d_vortex_y0 - position of vortex in xy plane
 * uint8_t d_Q_vortex - topological charge of vortex
 * NOTE: It is considered that x0 and y0 should be chosen out of lattice points in case 
 *       phase is corectly (mathematically) defined in every lattice point (check atan2).
 */
__global__ void __gpe_imprint_vortexline_zdir_(cuCplx *psi)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi = psi[ixyz];
    double abs_psi, phase;
    double _x,_y;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        
        _x = constgpu(ix) - 1.0*(NX/2) - d_vortex_x0;
        _y = constgpu(iy) - 1.0*(NY/2) - d_vortex_y0;
        
        abs_psi = hypot(lpsi.x, lpsi.y); //abs_psi = sqrt(lpsi.x*lpsi.x + lpsi.y*lpsi.y); (intrinsic should be faster)
        phase = atan2(_x,_y); // atan2(0,0) == -pi/2
        phase *= (double) (d_vortex_Q); //if (d_vortex_Q != 1) phase *= (double) (d_vortex_Q);
        lpsi.x = abs_psi*cos(phase);
        lpsi.y = abs_psi*sin(phase);
        
        psi[ixyz] = lpsi;
	    
    }
}

/*
 * This function imprints vortex parallel to z axis crossing x,y plane in (x0,y0) point
 * double d_vortex_x0, d_vortex_y0 - position of vortex in xy plane
 * uint8_t d_Q_vortex - topological charge of vortex
 * NOTE: It is considered that x0 and y0 should be chosen out of lattice points in case 
 *       phase is corectly (mathematically) defined in every lattice point (check atan2).
 */
__global__ void __gpe_imprint2_vortexline_zdir_(cuCplx *psi)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi = psi[ixyz];
    double abs_psi, phase;
    double _x,_y;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        
        _x = constgpu(ix) - 1.0*(NX/2) - d_vortex_x0;
        _y = constgpu(iy) - 1.0*(NY/2) - d_vortex_y0;
        
        abs_psi = hypot(lpsi.x, lpsi.y); //abs_psi = sqrt(lpsi.x*lpsi.x + lpsi.y*lpsi.y); (intrinsic should be faster)
        if (abs_psi > 1e-15)
        {
            phase = atan2(_x,_y); // atan2(0,0) == -pi/2
            phase *= (double) (d_vortex_Q); //if (d_vortex_Q != 1) phase *= (double) (d_vortex_Q);
            lpsi.x = abs_psi*cos(phase);
            lpsi.y = abs_psi*sin(phase);
            
            psi[ixyz] = lpsi;
	    }
    }
}

// ======================= More general enforcing phase =====================================================

__global__ void __gpe_compute_phase__(cuCplx* psi, double* d_phase)
{
	size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
	if(ixyz<nxyz)
    {
        d_phase[ixyz] = cplxArg(psi[ixyz]);
    }
}

int gpe_set_phase(double* h_phase)
{
	
    if (gpe_mem.d_phase == NULL) cuErrCheck( hipMalloc( &gpe_mem.d_phase, sizeof(double)*nxyz ) );
    
    cuErrCheck( hipMemcpy( gpe_mem.d_phase, h_phase, sizeof(double)*nxyz, hipMemcpyHostToDevice) ); 
    gpe_flags.phase_set=1;
    
    return GPE_SUCCESS;
}

int gpe_get_phase(double* h_phase)
{
	
    if (gpe_mem.d_phase == NULL) cuErrCheck( hipMalloc( &gpe_mem.d_phase, sizeof(double)*nxyz ) );
    
    __gpe_compute_phase__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi,gpe_mem.d_phase);
    if (h_phase) cuErrCheck( hipMemcpy( h_phase, gpe_mem.d_phase, sizeof(double)*nxyz, hipMemcpyDeviceToHost) );  // if h_phase != NULL copy phase to host
    gpe_flags.phase_set=1;
    
    return GPE_SUCCESS;
}

__global__ void __gpe_enforce_phase__(cuCplx* psi, double* d_phase)
{
	size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
	if(ixyz<nxyz)
    {
        psi[ixyz] = cplxScale( cplxExpi(d_phase[ixyz]), cplxAbs(psi[ixyz]) );
    }
}

// ======================= Density/Normalization ============================================================

// TODO: Test speed with cublas

/**
 * Function computes density from wave function psi
 * */
inline __device__  double gpe_density(cuCplx psi)
{
    return GAMMA * (psi.x*psi.x + psi.y*psi.y); // |psi|^2 * GAMMA, where GAMMA=1 for particles, GAMMA=2 for dimers
}

__global__ void __gpe_compute_density__(cuCplx *psi_in, double *rho_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        rho_out[ixyz] = gpe_density(psi_in[ixyz]);
    }
}

/**
 * Computes density and saves in array of complex numbers (as real part).
 * Suitable for dipolar interactions.
 * */
__global__ void __gpe_compute_density2C__(cuCplx *psi_in, cuCplx *rho_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        rho_out[ixyz].x = gpe_density(psi_in[ixyz]);
        rho_out[ixyz].y = 0.;
    }
}

__global__ void __gpe_normalize__(cuCplx *psi_inout, double *sumrho)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
//         if(ixyz==0) printf("sumrho[0]=%f\n", sumrho[0]);
        psi_inout[ixyz] = cplxScale(psi_inout[ixyz], sqrt(d_npart/sumrho[0]));
    }
}

// Normalizes wavefunction
int gpe_normalize(cuCplx *psi, double *wrk)
{
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk);
    cuErrCheck( local_reduction(wrk, nxyz, wrk, gpe_mem.threads, 0) );
    __gpe_normalize__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk);
    
    return GPE_SUCCESS;
}

int gpe_normalize_psi()
{
    return gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R);
}

static inline int gpe_normalize_psi(double *chemical_potential)
{    
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    cuErrCheck( local_reduction(gpe_mem.d_wrk2R, nxyz, gpe_mem.d_wrk2R, gpe_mem.threads, 0) );
    __gpe_normalize__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    if (chemical_potential)
    {
        double norm;
        cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R, sizeof(double), hipMemcpyDeviceToHost) ); 
        *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
    }
    
    return GPE_SUCCESS;
}

int gpe_get_density(double *t, double * density)
{
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    cuErrCheck( hipMemcpy( density , gpe_mem.d_wrk2R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );
    
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    return 0;
}


// =================== Accesing wavefunction =============================================================================

int gpe_set_psi(double t, cuCplx * psi)
{
    
    cuErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(d_t0), &t, sizeof(double)) ) ;
    gpe_mem.it=0;
    gpe_mem.t0=t;
    cuErrCheck( hipMemcpy( gpe_mem.d_psi , psi , sizeof(cuCplx)*nxyz, hipMemcpyHostToDevice ) );
    
    return 0;
}

int gpe_get_psi(double *t, cuCplx * psi)
{
    
    cuErrCheck( hipMemcpy( psi , gpe_mem.d_psi , sizeof(cuCplx)*nxyz, hipMemcpyDeviceToHost ) );
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    return 0;
}

// ======================= Evolution algorithm =============================================================================

/**
 * construct  exp(-i*dt*V/2) and apply exp(-i*dt*V/2) * psi 
 * */
__global__ void __gpe_exp_Vstep1_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        lrho = gpe_density(lpsi); // compute density
        lv=gpe_external_potential(ix, iy, iz, it) + gpe_dEDFdn(lrho,it); // external potential + mean field
        
        wrkR[ixyz]=lv; // it will be use later
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save
    }
}

__global__ void __gpe_exp_Vstep1_qf_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, double *qfpotential)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        lrho = gpe_density(lpsi); // compute density
        lv=gpe_external_potential(ix, iy, iz, it) + gpe_dEDFdn(lrho,it) + qfpotential[ixyz]; 
           // external potential + mean field + quantum friction potential
        
        wrkR[ixyz]=lv; // it will be use later
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save
    }    
}

__global__ void __gpe_exp_Vstep2_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        lpsi=cplxMul(lpsi, exp_lv); // finalize step from predictor
        
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1)); // external potential + mean field - take average
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep2_part1_(cuCplx *psi_in, cuCplx *psi_out, double * wrkR)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    
    // registers
    cuCplx exp_lv;

    if(ixyz<nxyz)
    {
        exp_lv = cplxExp( cplxScale(d_step_coeff, wrkR[ixyz]) );
        psi_out[ixyz]=cplxMul(psi_in[ixyz], exp_lv); // finalize step from predictor     
    }    
}

__global__ void __gpe_exp_Vstep2_part2_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1)); // external potential + mean field - take average
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep2_part2_qf_(uint it, cuCplx *psi_in, cuCplx *psi_out, double * wrkR, cuCplx * wrkC, double *qfpotential)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    // registers
    cuCplx lpsi, exp_lv;
    double lrho, lv;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        
        lpsi = psi_in[ixyz]; // psi to register
        lv = wrkR[ixyz]; // potentials to register
        lrho = gpe_density(lpsi); // compute density
        lv=0.5*(lv + gpe_external_potential(ix, iy, iz, it+1) + gpe_dEDFdn(lrho,it+1) + qfpotential[ixyz]) ; 
          // external potential + mean field + quantum friction potential - take average 
        exp_lv = cplxExp( cplxScale(d_step_coeff,lv) );
        wrkC[ixyz]=exp_lv; // it will be used later
        
        lpsi = psi_out[ixyz]; // psi to register
        lpsi=gpe_modify_psi(ix, iy, iz, it, lpsi); // modify psi
        psi_out[ixyz] = cplxMul(lpsi, exp_lv); // apply and save      
    }    
}

__global__ void __gpe_exp_Vstep3_(cuCplx *psi_inout, cuCplx * wrkC)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    
    if(ixyz<nxyz)
    {
        psi_inout[ixyz] = cplxMul(psi_inout[ixyz], wrkC[ixyz]); // apply and save      
    }    
}

__global__ void __gpe_multiply_by_expT__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    cuCplx _wavef;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        _wavef=psi_in[ixyz]; // bring to register
        _wavef=cplxMul(_wavef,d_exp_kkx2[ix]);
        _wavef=cplxMul(_wavef,d_exp_kky2[iy]);
        _wavef=cplxMul(_wavef,d_exp_kkz2_over_nxyz[iz]); // note - normalization factor is included here
        psi_out[ixyz]=_wavef; // send to global memory
    }    
}

__global__ void __gpe_multiply_by_k2_qf__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_qfcoeff*( d_kkx[ix]*d_kkx[ix] + d_kky[iy]*d_kky[iy] + d_kkz[iz]*d_kkz[iz] ) ); 
            
    }    
}

__global__ void __gpe_overlap_imag_qf__(cuCplx *psi1, cuCplx *psi2, double *overlap)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    double lrho;
    cuCplx lpsi;
    if(ixyz<nxyz)
    {
        lpsi = psi1[ixyz]; // psi to register
        lrho = gpe_density(lpsi); // compute density
        overlap[ixyz]= cplxMulI( cplxConj(lpsi),  psi2[ixyz] )/(lrho+GPE_QF_EPSILON);
    }
}

int gpe_compute_qf_potential(cuCplx *psi, cuCplx *wrk, double *qfpotential)
{
    hipfftResult cufft_result;
    
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, psi, wrk, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_k2_qf__<<<gpe_mem.blocks, gpe_mem.threads>>>(wrk, wrk);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, wrk, wrk, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_overlap_imag_qf__<<<gpe_mem.blocks, gpe_mem.threads>>>(psi, wrk, qfpotential);

        
    return 0;
}


// ===================== Evolution interface ===========================================

/**
 * Function evolves wave funcion nt steps 
 * */
int gpe_evolve(int nt)
{
    printf("Function not implemented!\n");
    return GPE_SUCCESS;
}

/**
 * Function evolves wave funcion nt steps 
 * */
int gpe_evolve_qf(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
                
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return 0;
}


/**
 * Simplest enforcing vortex phase with with method in both predictor and normal steps.
 */
int gpe_evolve_vortex(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        // changing the phase
        __gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
        cuErrCheck( hipGetLastError() );
        
        
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
                
                // changing the phase
                __gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
                cuErrCheck( hipGetLastError() );
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return GPE_SUCCESS;
}

/**
 * Enforces vortex phase with second method.
 */
int gpe_evolve_vortex2(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        // changing the phase
        __gpe_imprint2_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
        cuErrCheck( hipGetLastError() );
        
        
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
                
                // changing the phase
                __gpe_imprint2_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
                cuErrCheck( hipGetLastError() );
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return GPE_SUCCESS;
}

/**
 * Enforces vortex phase only in predictor step (first method).
 */
int gpe_evolve_vortex3(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        // changing the phase
        __gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
        cuErrCheck( hipGetLastError() );
        
        
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
                
                // changing the phase
                //__gpe_imprint_vortexline_zdir_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi);
                //cuErrCheck( hipGetLastError() );
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return GPE_SUCCESS;
}


// TODO: Check if this works better!
int gpe_evolve_enforced_phase(int nt, double* chemical_potential)
{
    hipfftResult cufft_result;
    int i;
        
    for(i=0; i<nt; i++)
    {
        // changing the phase
        __gpe_enforce_phase__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi,gpe_mem.d_phase);
        cuErrCheck( hipGetLastError() );
        
        
        if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
        {
            cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R) );
            __gpe_exp_Vstep1_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R, gpe_mem.d_wrk3R);
        }
        else
        {
            // potential part exp(V/2)
            __gpe_exp_Vstep1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        if(gpe_mem.beta==0.0 && gpe_mem.qfcoeff==0.0)
        {
            // without normalization
            __gpe_exp_Vstep2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
        }
        else
        {
            __gpe_exp_Vstep2_part1_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
            
            if(gpe_mem.beta!=0.0)
            {
                // with normalization between
                cuErrCheck( gpe_normalize(gpe_mem.d_psi2, gpe_mem.d_wrk2R+nxyz));
                
                // changing the phase
                __gpe_enforce_phase__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi,gpe_mem.d_phase);
                cuErrCheck( hipGetLastError() );
            }
            
            if(gpe_mem.qfcoeff!=0.0) // quantum friction is active
            {
                cuErrCheck( gpe_compute_qf_potential(gpe_mem.d_psi, gpe_mem.d_wrk3C, gpe_mem.d_wrk3R));
                __gpe_exp_Vstep2_part2_qf_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2, gpe_mem.d_wrk3R);
            }
            else
            {
                __gpe_exp_Vstep2_part2_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_psi2, gpe_mem.d_psi, gpe_mem.d_wrk2R, gpe_mem.d_psi2);
            }
        }
        
        // kinetic part exp(T)
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_FORWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        __gpe_multiply_by_expT__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi);
        cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi, HIPFFT_BACKWARD);
        if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
        
        // potential part exp(V/2)
        __gpe_exp_Vstep3_<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2);
        
        if(gpe_mem.beta!=0.0)
        {
            // normalize
            cuErrCheck( gpe_normalize(gpe_mem.d_psi, gpe_mem.d_wrk2R+nxyz));
            if (chemical_potential)
            {
                double norm;
                cuErrCheck( hipMemcpy( &norm, gpe_mem.d_wrk2R+nxyz, sizeof(double), hipMemcpyDeviceToHost) ); 
                *chemical_potential = -.5*log(norm/gpe_mem.npart)/gpe_mem.dt;
            } 
        }
        
        gpe_mem.it = gpe_mem.it + 1;
    }
    
    return GPE_SUCCESS;
}




// ========================= Energy counting =================================================

__global__ void __gpe_compute_vext__(uint it, double *rho, double *wrk)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        wrk[ixyz]=rho[ixyz]*gpe_external_potential(ix, iy, iz, it);
    }
}

__global__ void __gpe_compute_vint__(uint it, double *rho, double *wrk)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        wrk[ixyz]=gpe_EDF(rho[ixyz], it);
    }
}

__global__ void __gpe_compute_vext_vint__(uint it, double *rho, double *wrk1, double *wrk2)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    double lrho;
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 
        lrho=rho[ixyz];
        wrk1[ixyz]=lrho*gpe_external_potential(ix, iy, iz, it);
        wrk2[ixyz]=gpe_EDF(lrho, it);
    }
}

__global__ void __gpe_multiply_by_k2__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], ( d_kkx[ix]*d_kkx[ix] + d_kky[iy]*d_kky[iy] + d_kkz[iz]*d_kkz[iz] )/(constgpu(2.0*GAMMA*nxyz)) ); 
            
    }    
}

__global__ void __gpe_overlap_real__(cuCplx *psi1, cuCplx *psi2, double *overlap)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    if(ixyz<nxyz)
    {
        overlap[ixyz]= cplxMulR( cplxConj(psi1[ixyz]),  psi2[ixyz] );
    }
}

int gpe_energy(double *t, double *ekin, double *eint, double *eext)
{
    int ierr;
    hipfftResult cufft_result;
   
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    if(gpe_mem.beta==0.0) // normalize - otherwise is normalized every time step
    {
        ierr=gpe_normalize_psi();
        if(ierr!=0) return ierr;
    }
    
    // Compute density
    __gpe_compute_density__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2R);
    
    // Compute <V_ext> and <V_int>
    double * wrk1 = (double *)gpe_mem.d_wrk;
    double * wrk2 = wrk1 + nxyz;
    __gpe_compute_vext_vint__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.it, gpe_mem.d_wrk2R, wrk1, wrk2);
    cuErrCheck( local_reduction(wrk1, nxyz, wrk1, gpe_mem.threads, 0) );
    cuErrCheck( local_reduction(wrk2, nxyz, wrk2, gpe_mem.threads, 0) );
    cuErrCheck( hipMemcpy( eext , wrk1 , sizeof(double), hipMemcpyDeviceToHost ) );
    cuErrCheck( hipMemcpy( eint , wrk2 , sizeof(double), hipMemcpyDeviceToHost ) );
        
    // Compute <T> - kinetic energy
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi2, HIPFFT_FORWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    __gpe_multiply_by_k2__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_psi2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi2, gpe_mem.d_psi2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_psi2, gpe_mem.d_wrk2R);
    cuErrCheck( local_reduction(gpe_mem.d_wrk2R, nxyz, gpe_mem.d_wrk2R, gpe_mem.threads, 0) );
    cuErrCheck( hipMemcpy( ekin , gpe_mem.d_wrk2R , sizeof(double), hipMemcpyDeviceToHost ) );
    
    return 0;
}

// ========================== Currents of probability ================================

__global__ void __gpe_multiply_by_kx__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kkx[ix]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

__global__ void __gpe_multiply_by_ky__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kky[iy]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

__global__ void __gpe_multiply_by_kz__(cuCplx *psi_in, cuCplx *psi_out)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if(ixyz<nxyz)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i); 

        psi_out[ixyz]=cplxScale(psi_in[ixyz], d_kkz[iz]/(constgpu(GAMMA*nxyz)) ); 
    }    
}

int gpe_get_currents(double* t, double* jx, double* jy, double* jz)
{
    int ierr;
    
    hipfftResult cufft_result;
   
    *t = gpe_mem.t0 + gpe_mem.dt*gpe_mem.it;
    
    int alloc=0;
    if(gpe_mem.d_wrk3R==NULL) // I need extra memory
    {
        alloc=1;
        cuErrCheck( hipMalloc( &gpe_mem.d_wrk3R , sizeof(double)*nxyz ) );
    }
    
    // move to momentum space
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_psi, gpe_mem.d_psi2, HIPFFT_FORWARD);  
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;
    
    // Compute d / dx and jx
    __gpe_multiply_by_kx__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jx , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );    
    
    // Compute d / dy and jy
    __gpe_multiply_by_ky__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jy , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );
    
    // Compute d / dz and jz
    __gpe_multiply_by_kz__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi2, gpe_mem.d_wrk2);
    cufft_result=hipfftExecZ2Z(gpe_mem.plan, gpe_mem.d_wrk2, gpe_mem.d_wrk2, HIPFFT_BACKWARD);
    if(cufft_result!= HIPFFT_SUCCESS) return (int)cufft_result;    
    __gpe_overlap_real__<<<gpe_mem.blocks, gpe_mem.threads>>>(gpe_mem.d_psi, gpe_mem.d_wrk2, gpe_mem.d_wrk3R);    
    cuErrCheck( hipMemcpy( jz , gpe_mem.d_wrk3R , sizeof(double)*nxyz, hipMemcpyDeviceToHost ) );

    // Free memory
    if(alloc) 
    {
        cuErrCheck( hipFree(gpe_mem.d_wrk3R) );  
        gpe_mem.d_wrk3R = NULL;      
    }
    
    return GPE_SUCCESS;
}



// ======================================= TESTING ================================================================================

__global__ void print_gpu_array_nans( cuCplx* psi, int size)
{
    size_t ixyz= threadIdx.x + blockIdx.x * blockDim.x;
    uint ix, iy, iz, i;
    
    if( ixyz<nxyz && ixyz < size)
    {
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        ix -= nx/2;
        iy -= ny/2;
        iz -= nz/2;
        
        if (isnan(psi[i].x) || isnan(psi[i].y)) printf("x: %d\ty: %d\tz: %d\t\tpsi %e + %ej\n",ix,iy,iz,psi[i].x,psi[i].y);
    }
}

// ==================================================================================================================================
